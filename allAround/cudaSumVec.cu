﻿#include <iostream>
#include "hip/hip_runtime.h"


__global__ void vectorAdd(int N, double* d_a, double* d_b, double* d_c);
void* GPUalloc(void* hostp, std::size_t bytes);
void stop(const std::string error_message);

int main(){
	//set the size of the problem and the total amount of requsted memory
	constexpr int N = 1024;
	std::size_t bytes = sizeof(double) * N;

	//allocate memory on host
	double* h_a = new double[N];
	double* h_b = new double[N];
	double* h_c = new double[N];

	//initialize the two input arrays
	for (int ii = 0; ii < N; ++ii) {
		h_a[ii] = (double)ii;
		h_b[ii] = (double)ii;
		h_c[ii] = 0.0;
	}

	//allocate memory on device
	double* d_a = (double*)GPUalloc((void*)h_a, bytes);
	double* d_b = (double*)GPUalloc((void*)h_b, bytes);
	double* d_c = (double*)GPUalloc((void*)h_c, bytes);

	for (int ii = 0; ii < N; ++ii) {std::cout << h_a[ii] << "   " << h_b[ii] << "   " << h_c[ii] << '\n';}

	std::cout << "-------------------------------------------------------\n";
	
	//run CUDA kernel
	int numThreads = 128;
	int numBlocks = (N + numThreads - 1) / numThreads;

	std::cout << "Initiating CUDA kernel with: " << numBlocks << " blocks, and " << numThreads << " threads.\n";
	std::cout << "-------------------------------------------------------\n";
	std::cin.get();
	vectorAdd<<<numBlocks, numThreads>>> (N, d_a, d_b, d_c);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) stop("ERROR: CUDA KERNEL WAS NOT LAUNCHED SUCCESSFULLY\n");

	//copy resulted array from device to host
	err = hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	if (err != hipSuccess) stop("ERROR: COPY FROM DEVICE TO HOST WAS NOT SUCCESSFUL\n");

	//print the final result
	for (int ii = 0; ii < N; ++ii) {std::cout << h_c[ii] << '\n';}

	//deallocate memory on device
	hipFree(d_a); d_a = NULL;
	hipFree(d_b); d_b = NULL;
	hipFree(d_c); d_c = NULL;

	//deallocate memory on host
	if (h_a) { delete[] h_a; h_a = NULL; }
	if (h_b) { delete[] h_b; h_b = NULL; }
	if (h_c) { delete[] h_c; h_c = NULL; }

	std::cin.get();
	return 0;
}

__global__ void vectorAdd(int N, double* d_a, double* d_b, double* d_c) {
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadId < N){d_c[threadId] = d_a[threadId] + d_b[threadId];}
}

void* GPUalloc(void* hostp, std::size_t bytes){
	void* devp = NULL;
	hipError_t err1 = hipMalloc(&devp, bytes);
	if (err1 != hipSuccess) stop("ERROR: CUDA MEMORY ALLOCATION WAS NOT SUCCESSFUL -> TYPE 1");
		
	hipError_t err2 = hipMemcpy(devp, hostp, bytes, hipMemcpyHostToDevice);
	if (err2 != hipSuccess) stop("ERROR: CUDA MEMORY ALLOCATION WAS NOT SUCCESSFUL -> TYPE 2");

	return devp;
}

void stop(const std::string error_message){
	std::cerr << error_message << '\n';
	exit(1);
}