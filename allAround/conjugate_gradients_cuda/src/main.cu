#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <hip/hip_runtime.h>
#include "main.cuh"
#include "conjugateGradient.cuh"
#include "blocks.cuh"
#include "abort.cuh"
#include "allocDevMem.cuh"
#include ""


#define systemSize 1000
#define nthreads_reduction 128

int main(){
      //host/device allocations
      ConjugateGradientDataPtrs ptr;

      ptr.hostPtr = NULL; 
      allocHostMemorySpace(ptr.hostPtr);

      ptr.deviceData = NULL; 
      allocDeviceMemorySpace(ptr.deviceData);

      ptr.devicePtr = NULL;
      createDevicePtr(ptr.devicePtr, ptr.deviceData);

      //generate a linear system (Ax=b) with known solution
      formEqSystem(ptr);

      //solve with conjugate gradient
      constexpr int maximum_iterations = 10000;
      constexpr double stopping_criterior = -16;
      conjugatGradientSolver(ptr, maximum_iterations, stopping_criterior);

      //host/device deallocations
      freeHostMemory(ptr.hostPtr);
      freeDeviceMemory(ptr.deviceData);
      destroyDevicePtr(ptr.devicePtr);
}

// --------------------------------------------------------------------------------------
//                             H O S T  -  F U N C T I O N S
// --------------------------------------------------------------------------------------
void allocHostMemorySpace(ConjugateGradientData*& hostPtr){
      constexpr int nbThreads = nthreads_reduction;
      constexpr int nbBlocks  = getBlocks(nbThreads, systemSize);

      hostPtr = new ConjugateGradientData();

      hostPtr->mult1 = (double*)malloc(sizeof(double)); *(hostPtr->mult1) = 0.;
      hostPtr->mult2 = (double*)malloc(sizeof(double)); *(hostPtr->mult2) = 0.;

      hostPtr->alpha = (double*)malloc(sizeof(double)); *(hostPtr->alpha) = 0.;
      hostPtr->beta  = (double*)malloc(sizeof(double)); *(hostPtr->beta)  = 0.;

      hostPtr->projection = NULL;
      hostPtr->direction  = NULL;
      hostPtr->residual   = NULL;

      hostPtr->rhs     = NULL;
      hostPtr->diag    = NULL;
      hostPtr->offDiag = NULL;

      hostPtr->solution = (double*)malloc(systemSize*sizeof(double));
      hostPtr->aux      = (double*)malloc(nbBlocks  *sizeof(double));

      for (int i=0; i<systemSize; i++) {hostPtr->solution[i] = 0.;}
}

void allocDeviceMemorySpace(ConjugateGradientData*& deviceData){
      //allocate enough GPU memory for deviceData members
      deviceData = (ConjugateGradientData*)GPUalloc(sizeof(ConjugateGradientData), "deviceData");
}

void createDevicePtr(ConjugateGradientData*& devicePtr, ConjugateGradientData*& deviceData){
      devicePtr = (ConjugateGradientData*)GPUalloc(deviceData, sizeof(ConjugateGradientData), "devicePtr");
}

void freeHostMemory(ConjugateGradientData*& hostPtr){
      //deallocate hostPtr members from CPU memory
      if (hostPtr) free(hostPtr); hostPtr = NULL;
}

void freeDeviceMemory(ConjugateGradientData*& deviceData){
      //deallocate hostPtr members from GPU memory
      if (deviceData) hipFree(deviceData); deviceData = NULL;
}

void destroyDevicePtr(ConjugateGradientData*& devicePtr){
      if (devicePtr) hipFree(devicePtr); devicePtr = NULL;
}

void formEqSystem(ConjugateGradientDataPtrs& ptr){
      // Generate a tridiagonal Symmetric Positive Definite square matrix A :
      // --------------------------------------------------------------------
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      getSPD_GPU<<<nblck,nthr>>>(ptr.devicePtr);
      //check("getSPD_GPU failed");
}

void conjugatGradientSolver(ConjugateGradientDataPtrs& ptr, const int maximum_iterations, const double stopping_criterior){
      std::ofstream file("cg.conv", std::ios::out | std::ios::trunc);

      initSolution          (ptr);
      computeResidual       (ptr);
      initConjugateDirection(ptr);

      for (int iter=0; iter<maximum_iterations; iter++){
            matVec    (ptr, _Direction_, _Projection_);
            dotProduct(ptr, _Direction_, _Projection_, _Mult2_);
            dotProduct(ptr, _Residual_ , _Residual_  , _Mult1_);
            computeAlpha(ptr);

            const double residual = log10( sqrt(*ptr.hostPtr->mult1) + 1.E-21 );

            if (residual < stopping_criterior || iter%10 == 0 || iter == maximum_iterations-1){
                  std::cout << std::setw( 8) << iter     << " " << std::setw(15) << residual << std::endl;
            }

            file << std::setw( 8) << iter     << " " << std::setw(15) << residual << "\n";

            if (residual < stopping_criterior){
                  std::cout << "# Conjugate Gradient converged !!! \n"; break;
            }

            updateSolution(ptr);
            updateResidual(ptr);

            dotProduct (ptr, _Residual_, _Residual_, _Mult2_);
            computeBeta(ptr);

            updateConjugateDirection(ptr);
      }
      file.close();

      writeSolution(ptr);
}

void initSolution(ConjugateGradientDataPtrs& ptr){
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);
      
      initSolutionGPU<<<nblck,nthr>>>(ptr.devicePtr);
      //check("initSolutionGPU failed");
}

void computeResidual(ConjugateGradientDataPtrs& ptr){
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      computeResidualGPU<<<nblck,nthr>>>(ptr.devicePtr);
      //check("computeResidualGPU failed");
}

void initConjugateDirection(ConjugateGradientDataPtrs& ptr){
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      initConjugateDirectionGPU<<<nblck,nthr>>>(ptr.devicePtr);
      //check("initConjugateDirectionGPU failed");
}

void dotProduct(ConjugateGradientDataPtrs& ptr, ConjugateGradientEnum A, ConjugateGradientEnum B, ConjugateGradientEnum C){
      //     Ipologiste to esoteriko ginomeno ton dianismaton A kai B. To apotelesma na 
      // filaxthei sti scalar metabliti C pou prepei gnorizoun toso i CPU oso kai i GPU.
      // 
    const int nthr = 128;
    const int nblck = getBlocks(nthr, systemSize);

    matVec();
      //
      // B O I T H E I A :
      //
      //   ta "ptr.hostPtr->get(A)", "ptr.hostPtr->get(B)" kai "ptr.hostPtr->get(C)" sas
      // girizoun tous CPU pointers ton dianismaton A kai B kai tis scalar posotitas C
      // antistoixa.
      //
      //   omoia ta "ptr.deviceData->get(A)", "ptr.deviceData->get(B)" kai "ptr.deviceData->get(C)"
      // sas girizoun tous antistoixous GPU pointers. Pointers diladi pou deixnoun se theseis
      // mnimis tis GPU
      //
}

void matVec(ConjugateGradientDataPtrs& ptr, ConjugateGradientEnum A, ConjugateGradientEnum B){
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      matVecGPU<<<nblck,nthr>>>(ptr.deviceData->get(_Diag_), ptr.deviceData->get(_OffDiag_), ptr.deviceData->get(A), ptr.deviceData->get(B));
      //check("matVecGPU failed");
}

void computeAlpha(ConjugateGradientDataPtrs& ptr){
      *ptr.hostPtr->alpha = *ptr.hostPtr->mult1 / *ptr.hostPtr->mult2;
}

void computeBeta(ConjugateGradientDataPtrs& ptr){
      *ptr.hostPtr->beta = *ptr.hostPtr->mult2 / *ptr.hostPtr->mult1;
}

void updateSolution(ConjugateGradientDataPtrs& ptr){
      const int nthr  = 128;
      const int nblck = getBlocks(nthr, systemSize);

      updateSolutionGPU<<<nblck,nthr>>>(ptr.devicePtr, *ptr.hostPtr->alpha);
      //check("updateSolutionGPU failed");
}

void updateResidual(ConjugateGradientDataPtrs& ptr){
      const int nthr  = 128;
      const int nblck = getBlocks(nthr, systemSize);

      updateResidualGPU<<<nblck,nthr>>>(ptr.devicePtr, *ptr.hostPtr->alpha);
      //check("updateResidualGPU failed");
}

void updateConjugateDirection(ConjugateGradientDataPtrs& ptr){
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      updateConjugateDirectionGPU<<<nblck,nthr>>>(ptr.devicePtr, *ptr.hostPtr->beta);
      //check("updateConjugateDirectionGPU failed");
}

void writeSolution(ConjugateGradientDataPtrs& ptr){
      std::ofstream file;
      file.open("cg.res",std::ios::out | std::ios::trunc);

      hipError_t err = hipMemcpy(ptr.hostPtr->solution, ptr.deviceData->solution, systemSize*sizeof(double), hipMemcpyDeviceToHost);
      if (err != hipSuccess) Stop("memory copy failed");

      for (int i=0; i<systemSize; i++){
            file << std::setw(15) << ptr.hostPtr->solution[i] << "\n";
      }

      file.close();
}


// --------------------------------------------------------------------------------------
//                                    K E R N E L S
// --------------------------------------------------------------------------------------
__global__ void getSPD_GPU(ConjugateGradientData* data){
      const int index = blockIdx.x*blockDim.x + threadIdx.x;
      if (index >= systemSize) return;

      constexpr double h    = 0.05;
      constexpr double T0   = 300.0;
      constexpr double Tinf = 200.0;
      constexpr double Tend = 400.0;
      constexpr double len  =  10.0;

      constexpr double dx   = len / double(systemSize-1);
      constexpr double fac  = h*dx*dx;

      // initialize solution :
      //double sol = 0.;
      //if (index == 0)            sol = T0;   // Dirichlet condition
      //if (index == systemSize-1) sol = Tend; // Dirichlet condition

      //data->solution[index] = sol;

      // LHS / RHS :
      double rhs     = Tinf* fac;
      double diag    = 2.  + fac;
      double offDiag =-1.;

      if (index == 0           ) { rhs = T0;   diag = 1.; }
      if (index == systemSize-1) { rhs = Tend; diag = 1.; }

      data->rhs [index] = rhs;
      data->diag[index] = diag;

      if (index < systemSize-1) data->offDiag[index] = offDiag;
}

__global__ void initSolutionGPU(ConjugateGradientData* data){
      //initialize solution vector to zero except for the boundaries
      const int index = blockIdx.x * blockDim.x + threadIdx.x;

      constexpr double T0 = 300.0;
      constexpr double Tend = 400.0;

      double sol = 0.0;

      if (index == 0) sol = T0;
      if (index == systemSize - 1) sol = Tend;

      data->solution[index] = sol;
}

__global__ void initConjugateDirectionGPU(ConjugateGradientData* data){
      //initialize the conjugate gradient direction
      const int index = blockIdx.x * blockDim.x + threadIdx.x;

      //initially the conjugate direction is equal to the initial residual: p0 = r0 = b0 - A * x0
      computeResidualGPU(data);
      data->direction[index] = data->residual[index];
}

__global__ void computeResidualGPU(ConjugateGradientData* data){
      const int index = blockIdx.x * blockDim.x + threadIdx.x;
      //residual = rhs - A*solution
      matVecGPU(data->diag, data->offDiag, data->solution, data->projection);
      data->residual[index] = data->rhs[index] - data->projection[index];
}

__global__ void matVecGPU(double* diag, double* offDiag, double* x, double* y){
      //  y = A*x
      // Ipologiste to ginomeno A*x me y to apotelesma tou ipologismou
}

__global__ void updateSolutionGPU(ConjugateGradientData* data, const double alpha){
      // Ananeoste ti lisi ("solution")
}

__global__ void updateResidualGPU(ConjugateGradientData* data, const double alpha){
      // Ananeoste to residual
}

__global__ void updateConjugateDirectionGPU(ConjugateGradientData* data, const double beta){
      // Ananeoste tis conjugate directions ("direction")
}
