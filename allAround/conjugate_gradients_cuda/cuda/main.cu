#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <iomanip>
#include <fstream>

#include "main.cuh"
#include "conjugateGradient.cuh"

// ... INCLUDE REST HEADERS ... 

#define systemSize 1000 // The order of linear system
#define nthreads_reduction 128

//
// **************************************************************************************
int main()
// **************************************************************************************
{
      // host/device allocations :
      ConjugateGradientDataPtrs ptr;

      ptr.hostPtr    = NULL; allocHostMemorySpace  (ptr.hostPtr   );
      ptr.deviceData = NULL; allocDeviceMemorySpace(ptr.deviceData);
      ptr.devicePtr  = NULL; createDevicePtr(ptr.devicePtr, ptr.deviceData);

      // Generate a linear system (Ax=b) with known solution :
      formEqSystem(ptr);

      // Solve with Conjugate Gradient :
      const int    maximum_iterations = 10000;
      const double stopping_criterior =-16;
      conjugatGradientSolver(ptr, maximum_iterations,stopping_criterior);

      // host/device deallocations :
      freeHostMemory  (ptr.hostPtr   );
      freeDeviceMemory(ptr.deviceData);
      destroyDevicePtr(ptr.devicePtr );
}


//
// --------------------------------------------------------------------------------------
//                             H O S T  -  F U N C T I O N S
// --------------------------------------------------------------------------------------
//


//
// **************************************************************************************
void allocHostMemorySpace(ConjugateGradientData*& hostPtr)
// **************************************************************************************
{
      const int nbThreads = nthreads_reduction;
      const int nbBlocks  = getBlocks(nbThreads,systemSize);

      hostPtr = new ConjugateGradientData();

      hostPtr->mult1 = (double*)malloc(sizeof(double)); *hostPtr->mult1 = 0.;
      hostPtr->mult2 = (double*)malloc(sizeof(double)); *hostPtr->mult2 = 0.;

      hostPtr->alpha = (double*)malloc(sizeof(double)); *hostPtr->alpha = 0.;
      hostPtr->beta  = (double*)malloc(sizeof(double)); *hostPtr->beta  = 0.;

      hostPtr->projection = NULL;
      hostPtr->direction  = NULL;
      hostPtr->residual   = NULL;

      hostPtr->rhs        = NULL;
      hostPtr->diag       = NULL;
      hostPtr->offDiag    = NULL;

      hostPtr->solution   = (double*)malloc(systemSize*sizeof(double));
      hostPtr->aux        = (double*)malloc(nbBlocks  *sizeof(double));

      for (int i=0; i<systemSize; i++)
      {
            hostPtr->solution[i] = 0.;
      }
}

//
// **************************************************************************************
void allocDeviceMemorySpace(ConjugateGradientData*& deviceData)
// **************************************************************************************
{
      // Desmeuste arketo xoro sti mnimi tis GPU gia ta members tou deviceData
      // xrisimopoiontas tin function "GPUalloc"
}

//
// **************************************************************************************
void createDevicePtr
(
      ConjugateGradientData*& devicePtr,
      ConjugateGradientData*& deviceData
)
// **************************************************************************************
{
      devicePtr = (ConjugateGradientData*)GPUalloc(deviceData,sizeof(ConjugateGradientData),"devicePtr");
}

//
// **************************************************************************************
void freeHostMemory(ConjugateGradientData*& hostPtr)
// **************************************************************************************
{
      // Apodesmeuste ton xoro mnimis pou pianoun ta members tou hostPtr
      // xrisimopoiontas tin function "free"
}

//
// **************************************************************************************
void freeDeviceMemory(ConjugateGradientData*& deviceData)
// **************************************************************************************
{
      // Apodesmeuste ton xoro mnimis pou pianoun ta members tou hostPtr
      // xrisimopoiontas tin function "hipFree"
}

//
// **************************************************************************************
void destroyDevicePtr(ConjugateGradientData*& devicePtr)
// **************************************************************************************
{
      if (devicePtr) hipFree(devicePtr); devicePtr = NULL;
}

//
// **************************************************************************************
void formEqSystem(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      // Generate a tridiagonal Symmetric Positive Definite square matrix A :
      // --------------------------------------------------------------------
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      getSPD_GPU<<<nblck,nthr>>>(ptr.devicePtr);
      check("getSPD_GPU failed");
}

//
// **************************************************************************************
void conjugatGradientSolver
(
      ConjugateGradientDataPtrs& ptr, const int maximum_iterations, const double stopping_criterior
)
// **************************************************************************************
{
      std::ofstream file("cg.conv", std::ios::out | std::ios::trunc);

      initSolution          (ptr);
      computeResidual       (ptr);
      initConjugateDirection(ptr);

      for (int iter=0; iter<maximum_iterations; iter++)
      {
            matVec    (ptr, _Direction_, _Projection_);
            dotProduct(ptr, _Direction_, _Projection_, _Mult2_);
            dotProduct(ptr, _Residual_ , _Residual_  , _Mult1_);
            computeAlpha(ptr);

            const double residual = log10( sqrt(*ptr.hostPtr->mult1) + 1.E-21 );

            if (residual < stopping_criterior || iter%10 == 0 || iter == maximum_iterations-1)
            {
                  std::cout << std::setw( 8) << iter     << " "
                            << std::setw(15) << residual << std::endl;
            }

            file << std::setw( 8) << iter     << " "
                 << std::setw(15) << residual << "\n";

            if (residual < stopping_criterior)
            {
                  std::cout << "# Conjugate Gradient converged !!! \n"; break;
            }

            updateSolution(ptr);
            updateResidual(ptr);

            dotProduct (ptr, _Residual_, _Residual_, _Mult2_);
            computeBeta(ptr);

            updateConjugateDirection(ptr);
      }
      file.close();

      writeSolution(ptr);
}

//
// **************************************************************************************
void initSolution(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);
      
      initSolutionGPU<<<nblck,nthr>>>(ptr.devicePtr);
      check("initSolutionGPU failed");
}

//
// **************************************************************************************
void computeResidual(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      computeResidualGPU<<<nblck,nthr>>>(ptr.devicePtr);
      check("computeResidualGPU failed");
}

//
// **************************************************************************************
void initConjugateDirection(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      initConjugateDirectionGPU<<<nblck,nthr>>>(ptr.devicePtr);
      check("initConjugateDirectionGPU failed");
}

//
// **************************************************************************************
void dotProduct
(
      ConjugateGradientDataPtrs& ptr,
      ConjugateGradientEnum A,
      ConjugateGradientEnum B,
      ConjugateGradientEnum C
)
// **************************************************************************************
{
      //     Ipologiste to esoteriko ginomeno ton dianismaton A kai B. To apotelesma na 
      // filaxthei sti scalar metabliti C pou prepei gnorizoun toso i CPU oso kai i GPU.
      //
      //
      // B O I T H E I A :
      //
      //   ta "ptr.hostPtr->get(A)", "ptr.hostPtr->get(B)" kai "ptr.hostPtr->get(C)" sas
      // girizoun tous CPU pointers ton dianismaton A kai B kai tis scalar posotitas C
      // antistoixa.
      //
      //   omoia ta "ptr.deviceData->get(A)", "ptr.deviceData->get(B)" kai "ptr.deviceData->get(C)"
      // sas girizoun tous antistoixous GPU pointers. Pointers diladi pou deixnoun se theseis
      // mnimis tis GPU
      //
}

//
// **************************************************************************************
void matVec
(
      ConjugateGradientDataPtrs& ptr,
      ConjugateGradientEnum A,
      ConjugateGradientEnum B
)
// **************************************************************************************
{
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      matVecGPU<<<nblck,nthr>>>
      (
            ptr.deviceData->get(_Diag_   ),
            ptr.deviceData->get(_OffDiag_),
            ptr.deviceData->get(A),
            ptr.deviceData->get(B)
      );
      check("matVecGPU failed");
}

//
// **************************************************************************************
void computeAlpha(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      // compute alpha :
      *ptr.hostPtr->alpha = *ptr.hostPtr->mult1 / *ptr.hostPtr->mult2;
}

//
// **************************************************************************************
void computeBeta(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      // compute beta :
      *ptr.hostPtr->beta = *ptr.hostPtr->mult2 / *ptr.hostPtr->mult1;
}

//
// **************************************************************************************
void updateSolution(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      updateSolutionGPU<<<nblck,nthr>>>(ptr.devicePtr, *ptr.hostPtr->alpha);
      check("updateSolutionGPU failed");
}

//
// **************************************************************************************
void updateResidual(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      updateResidualGPU<<<nblck,nthr>>>(ptr.devicePtr, *ptr.hostPtr->alpha);
      check("updateResidualGPU failed");
}

//
// **************************************************************************************
void updateConjugateDirection(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      const int nthr  = 128;
      const int nblck = getBlocks(nthr,systemSize);

      updateConjugateDirectionGPU<<<nblck,nthr>>>(ptr.devicePtr, *ptr.hostPtr->beta);
      check("updateConjugateDirectionGPU failed");
}

//
// **************************************************************************************
void writeSolution(ConjugateGradientDataPtrs& ptr)
// **************************************************************************************
{
      std::ofstream file;
      file.open("cg.res",std::ios::out | std::ios::trunc);

      hipError_t err = hipMemcpy
                        (
                              ptr.hostPtr   ->solution, 
                              ptr.deviceData->solution, systemSize*sizeof(double), hipMemcpyDeviceToHost
                        );
      if (err != hipSuccess) Stop("memory copy failed");

      for (int i=0; i<systemSize; i++)
      {
            file << std::setw(15) << ptr.hostPtr->solution[i] << "\n";
      }

      file.close();
}


//
// --------------------------------------------------------------------------------------
//                                    K E R N E L S
// --------------------------------------------------------------------------------------
//


//
// **************************************************************************************
__global__ void getSPD_GPU(ConjugateGradientData* data)
// **************************************************************************************
{
      const int index = blockIdx.x*blockDim.x + threadIdx.x;
      if (index >= systemSize) return;

      const double h    = 0.05;
      const double T0   = 300.0;
      const double Tinf = 200.0;
      const double Tend = 400.0;
      const double len  =  10.0;

      const double dx   = len / double(systemSize-1);
      const double fac  = h*dx*dx;

      // initialize solution :
      double sol = 0.;
      if (index == 0           ) sol = T0;   // Dirichlet condition
      if (index == systemSize-1) sol = Tend; // Dirichlet condition

      data->solution[index] = sol;

      // LHS / RHS :
      double rhs     = Tinf* fac;
      double diag    = 2.  + fac;
      double offDiag =-1.;

      if (index == 0           ) { rhs = T0;   diag = 1.; }
      if (index == systemSize-1) { rhs = Tend; diag = 1.; }

      data->rhs [index] = rhs;
      data->diag[index] = diag;

      if (index < systemSize-1) data->offDiag[index] = offDiag;
}

//
// **************************************************************************************
__global__ void initSolutionGPU(ConjugateGradientData* data)
// **************************************************************************************
{
      // Arxikopoiiste to dianisma tis lisis ("solution") me 0 ektos ton orion
}

//
// **************************************************************************************
__global__ void initConjugateDirectionGPU(ConjugateGradientData* data)
// **************************************************************************************
{
      // Arxikopoiiste tin conjugate direction ("direction")
}

//
// **************************************************************************************
__global__ void computeResidualGPU(ConjugateGradientData* data)
// **************************************************************************************
{ //  residual = rhs - A*solution
      // Ipologiste to residual
}

//
// **************************************************************************************
__global__ void matVecGPU(double* diag, double* offDiag, double* x, double* y)
// **************************************************************************************
{ //  y = A*x
      // Ipologiste to ginomeno A*x me y to apotelesma tou ipologismou
}

//
// **************************************************************************************
__global__ void updateSolutionGPU(ConjugateGradientData* data, const double alpha)
// **************************************************************************************
{
      // Ananeoste ti lisi ("solution")
}

//
// **************************************************************************************
__global__ void updateResidualGPU(ConjugateGradientData* data, const double alpha)
// **************************************************************************************
{
      // Ananeoste to residual
}

//
// **************************************************************************************
__global__ void updateConjugateDirectionGPU(ConjugateGradientData* data, const double beta)
// **************************************************************************************
{
      // Ananeoste tis conjugate directions ("direction")
}

// rest host/device functions :
// _____________________________________________________________
//
// .... INCLUDE REST SOURCE FILES ....
