#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>

__global__ void printThreadId();

int main() {
    std::cout << "Hello\n";

    printThreadId << <2, 16 >> > ();

    hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "something went wrong...\n";
        exit(1);
    }

    std::cin.get();

    return 0;
}

__global__ void printThreadId() {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    printf("# hello world from thread %d of block %d and id %d\n", threadIdx.x, blockIdx.x, threadId);
}