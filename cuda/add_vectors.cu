﻿#include <iostream>
#include <stdio.h>
#include <hipblas.h>
#include "hip/hip_runtime.h"


// Forward declarations of GPU kernels
__global__ void GPUprintThreadId();
__global__ void GPUvectorAdd(int N, float *d_a, float *d_b, float *d_c);

// Forward declarations of host functions
void *allocGPUmemory(void *hostp, std::size_t bytes);
void stop(const std::string &error_message);

int main() {
  // Set the size of the problem and the total amount of required memory
  constexpr int N = 1 << 26;

  std::cout << "vector size is equal to: " << N << '\n';
  std::size_t bytes = sizeof(float) * N;

  // Allocate host memory
  float *h_a     = new float[N];
  float *h_b     = new float[N];
  float *h_c     = new float[N];
  float *h_cblas = new float[N];

  // Initialize vectors
  for (int ii = 0; ii < N; ++ii) {
    h_a[ii]     = static_cast<float>(ii);
    h_b[ii]     = static_cast<float>(ii);
    h_c[ii]     = 0.0f;
    h_cblas[ii] = 0.0f;
  }

  // Allocate device memory
  float *d_a = static_cast<float *>(allocGPUmemory(static_cast<void *>(h_a), bytes));
  float *d_b = static_cast<float *>(allocGPUmemory(static_cast<void *>(h_b), bytes));
  float *d_c = static_cast<float *>(allocGPUmemory(static_cast<void *>(h_c), bytes));

  // Initialize and run CUDA kernel
  int numThreads = 32;
  int numBlocks = (N + numThreads - 1) / numThreads;

  std::cout << "Initiating CUDA kernel with: " << numBlocks << " blocks, and " << numThreads << " threads.\n";
  std::cout << "-------------------------------------------------------\n";
  std::cin.get();

  // Deploy custom vector addition kernel
  std::cout << "Running custom vector addition kernel\n";
  //GPUprintThreadId<<<numBlocks, numThreads>>>();
  GPUvectorAdd<<<numBlocks, numThreads>>>(N, d_a, d_b, d_c);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) stop("ERROR: CUDA KERNEL WAS NOT LAUNCHED SUCCESSFULLY\n");

  // Copy the resulting array from device back to the host
  err = hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
  if (err != hipSuccess) stop("ERROR: COPY FROM DEVICE TO HOST WAS NOT SUCCESSFUL\n");

  // Deploy cublas saxpy kernel
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSetVector(N, sizeof(float), h_a, 1, d_a, 1);
  hipblasSetVector(N, sizeof(float), h_a, 1, d_a, 1);

  const float scale = 1.0;

  hipblasSaxpy(handle, N, &scale, (float *)(d_a), 1, (float *)(d_b), 1);

  hipblasGetVector(N, sizeof(float), d_b, 1, h_cblas, 1);

  hipblasDestroy(handle);

  float maxError = 0.0;
  for (int ii = 0; ii < N; ++ii) {
    maxError = std::max(maxError, abs(h_c[ii] - h_cblas[ii]));
    //std::cout << h_c[ii] << "  " << h_cblas[ii] << ", Max error: " << maxError << '\n';
  }

  if (maxError < 1.0e-5) {
    std::cout << "Program executed successfully\n";
  } else {
    std::cout << "Something went wrong\n";
  }

  // Deallocate device memory
  hipFree(d_a); d_a = NULL;
  hipFree(d_b); d_b = NULL;
  hipFree(d_c); d_c = NULL;

  // Deallocate host memory
  if (h_a)     { delete[] h_a; h_a = NULL; }
  if (h_b)     { delete[] h_b; h_b = NULL; }
  if (h_c)     { delete[] h_c; h_c = NULL; }
  if (h_cblas) { delete[] h_cblas; h_cblas = NULL; }

  std::cout << "Program finished\n";
  std::cin.get();

  return 0;
}

// Kernel definitions
__global__ void GPUvectorAdd(int N, float *d_a, float *d_b, float *d_c) {
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadId < N) {
    d_c[threadId] = d_a[threadId] + d_b[threadId];
  }
}

__global__ void GPUprintThreadId() {
  printf(" blockID:  %d, threadID: %d\n", blockIdx.x, threadIdx.x);
}

// Host function definitions
void *allocGPUmemory(void *hostp, std::size_t bytes) {
  void* devp = NULL;
  hipError_t err1 = hipMalloc(&devp, bytes);
  if (err1 != hipSuccess) stop("ERROR: CUDA MEMORY ALLOCATION WAS NOT SUCCESSFUL -> TYPE 1");

  hipError_t err2 = hipMemcpy(devp, hostp, bytes, hipMemcpyHostToDevice);
  if (err2 != hipSuccess) stop("ERROR: CUDA MEMORY ALLOCATION WAS NOT SUCCESSFUL -> TYPE 2");

  return devp;
}

void stop(const std::string &error_message) {
  std::cerr << error_message << '\n';
  exit(1);
}
