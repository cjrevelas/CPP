﻿#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"


// Forward declarations of GPU kernels
__global__ void GPUprintThreadId();
__global__ void GPUvectorAdd(int N, double *d_a, double *d_b, double *d_c);

// Forward declarations of host functions
void *allocGPUmemory(void *hostp, std::size_t bytes);
void stop(const std::string &error_message);


int main() {
	// Set the size of the problem and the total amount of required memory
	constexpr int N = 128;
	std::size_t bytes = sizeof(double) * N;

	// Allocate host memory
	double *h_a = new double[N];
	double *h_b = new double[N];
	double *h_c = new double[N];

	// Initialize the two input arrays
	for (int ii = 0; ii < N; ++ii) {
		h_a[ii] = static_cast<double>(ii);
		h_b[ii] = static_cast<double>(ii);
		h_c[ii] = 0.0;
	}

	// Allocate device memory
	double* d_a = static_cast<double *>(allocGPUmemory(static_cast<void *>(h_a), bytes));
	double* d_b = static_cast<double *>(allocGPUmemory(static_cast<void *>(h_b), bytes));
	double* d_c = static_cast<double *>(allocGPUmemory(static_cast<void *>(h_c), bytes));

	// Initialize and run CUDA kernel
	int numThreads = 32;
	int numBlocks = (N + numThreads - 1) / numThreads;

	std::cout << "Initiating CUDA kernel with: " << numBlocks << " blocks, and " << numThreads << " threads.\n";
	std::cout << "-------------------------------------------------------\n";
	std::cin.get();

	// Deploy kernels
	GPUprintThreadId<<<numBlocks, numThreads>>>();

	GPUvectorAdd<<<numBlocks, numThreads>>>(N, d_a, d_b, d_c);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) stop("ERROR: CUDA KERNEL WAS NOT LAUNCHED SUCCESSFULLY\n");

	// Copy resulted array from device to host
	err = hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	if (err != hipSuccess) stop("ERROR: COPY FROM DEVICE TO HOST WAS NOT SUCCESSFUL\n");

	// Print the final result
	for (int ii = 0; ii < N; ++ii) {
		std::cout << h_c[ii] << '\n';
	}

	// Deallocate device memory
	hipFree(d_a); d_a = NULL;
	hipFree(d_b); d_b = NULL;
	hipFree(d_c); d_c = NULL;

	// Deallocate host memory 
	if (h_a) { delete[] h_a; h_a = NULL; }
	if (h_b) { delete[] h_b; h_b = NULL; }
	if (h_c) { delete[] h_c; h_c = NULL; }

	std::cin.get();

	return 0;
}


// Kernel definitions
__global__ void GPUvectorAdd(int N, double *d_a, double *d_b, double *d_c) {
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadId < N) {
		d_c[threadId] = d_a[threadId] + d_b[threadId];
	}
}


__global__ void GPUprintThreadId() {
	printf(" blockID:  %d, threadID: %d\n", blockIdx.x, threadIdx.x);
}


// Host function definitions
void *allocGPUmemory(void *hostp, std::size_t bytes) {
	void* devp = NULL;
	hipError_t err1 = hipMalloc(&devp, bytes);
	if (err1 != hipSuccess) stop("ERROR: CUDA MEMORY ALLOCATION WAS NOT SUCCESSFUL -> TYPE 1");

	hipError_t err2 = hipMemcpy(devp, hostp, bytes, hipMemcpyHostToDevice);
	if (err2 != hipSuccess) stop("ERROR: CUDA MEMORY ALLOCATION WAS NOT SUCCESSFUL -> TYPE 2");

	return devp;
}


void stop(const std::string &error_message) {
	std::cerr << error_message << '\n';
	exit(1);
}
