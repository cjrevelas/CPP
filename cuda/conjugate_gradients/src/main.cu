#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <hip/hip_runtime.h>
#include "main.cuh"
#include "conjugateGradient.cuh"
#include "blocks.cuh"
#include "abort.cuh"
#include "allocDevMem.cuh"
#include ""

#define systemSize 1000
#define nthreads_reduction 128

int main() {
  // Host/device allocations
  ConjugateGradientDataPtrs ptr;

  ptr.hostPtr = NULL;
  allocHostMemorySpace(ptr.hostPtr);

  ptr.deviceData = NULL;
  allocDeviceMemorySpace(ptr.deviceData);

  ptr.devicePtr = NULL;
  createDevicePtr(ptr.devicePtr, ptr.deviceData);

  // Generate a linear system (Ax=b) with known solution
  formEqSystem(ptr);

  // Solve with conjugate gradient
  constexpr int maxIter = 10000;
  constexpr double tolerance = -16;
  conjugatGradientSolver(ptr, maxIter, tolerance);

  // Host/device deallocations
  freeHostMemory(ptr.hostPtr);
  freeDeviceMemory(ptr.deviceData);
  destroyDevicePtr(ptr.devicePtr);
}

// --------------------------------------------------------------------------------------
//                             H O S T  -  F U N C T I O N S
// --------------------------------------------------------------------------------------
void allocHostMemorySpace(ConjugateGradientData*& hostPtr) {
  constexpr int nbThreads = nthreads_reduction;
  constexpr int nbBlocks  = getBlocks(nbThreads, systemSize);

  hostPtr = new ConjugateGradientData();

  hostPtr->mult1 = (double*)malloc(sizeof(double));
  hostPtr->mult2 = (double*)malloc(sizeof(double));

  *(hostPtr->mult1) = 0.;
  *(hostPtr->mult2) = 0.;

  hostPtr->alpha = (double*)malloc(sizeof(double));
  hostPtr->beta  = (double*)malloc(sizeof(double));

  *(hostPtr->alpha) = 0.;
  *(hostPtr->beta)  = 0.;

  hostPtr->projection = NULL;
  hostPtr->direction  = NULL;
  hostPtr->residual   = NULL;

  hostPtr->rhs     = NULL;
  hostPtr->diag    = NULL;
  hostPtr->offDiag = NULL;

  hostPtr->solution = (double*)malloc(systemSize * sizeof(double));
  hostPtr->aux      = (double*)malloc(nbBlocks   * sizeof(double));

  for (int i=0; i<systemSize; i++) { hostPtr->solution[i] = 0.; }
}

void allocDeviceMemorySpace(ConjugateGradientData *&deviceData) {
  // Allocate enough GPU memory for deviceData members
  deviceData = (ConjugateGradientData*)GPUalloc(sizeof(ConjugateGradientData), "deviceData");
}

void createDevicePtr(ConjugateGradientData *&devicePtr, ConjugateGradientData *&deviceData) {
  devicePtr = (ConjugateGradientData*)GPUalloc(deviceData, sizeof(ConjugateGradientData), "devicePtr");
}

void freeHostMemory(ConjugateGradientData *&hostPtr) {
  // Deallocate hostPtr members from CPU memory
  if (hostPtr) free(hostPtr); hostPtr = NULL;
}

void freeDeviceMemory(ConjugateGradientData*& deviceData) {
  // Deallocate hostPtr members from GPU memory
  if (deviceData) hipFree(deviceData); deviceData = NULL;
}

void destroyDevicePtr(ConjugateGradientData*& devicePtr) {
  if (devicePtr) hipFree(devicePtr); devicePtr = NULL;
}

void formEqSystem(ConjugateGradientDataPtrs& ptr) {
  // Generate a tridiagonal Symmetric Positive Definite square matrix A :
  // --------------------------------------------------------------------
  const int nthr  = 128;
  const int nblck = getBlocks(nthr,systemSize);

  getSPD_GPU<<<nblck,nthr>>>(ptr.devicePtr);
  //check("getSPD_GPU failed");
}

void conjugatGradientSolver(ConjugateGradientDataPtrs &ptr, const int maxIter, const double tolerance) {
  std::ofstream file("cg.conv", std::ios::out | std::ios::trunc);

  initSolution(ptr);

  computeResidual(ptr);

  initConjugateDirection(ptr);

  for (int iter=0; iter<maxIter; iter++) {
    matVec    (ptr, _Direction_, _Projection_);

    dotProduct(ptr, _Direction_, _Projection_, _Mult2_);

    dotProduct(ptr, _Residual_ , _Residual_  , _Mult1_);

    computeAlpha(ptr);

    const double residual = log10( sqrt(*ptr.hostPtr->mult1) + 1.E-21 );

    if (residual < tolerance || iter%10 == 0 || iter == maxIter-1) {
      std::cout << std::setw( 8) << iter << ' ' << std::setw(15) << residual << '\n';
    }

    file << std::setw( 8) << iter << ' ' << std::setw(15) << residual << '\n';

    if (residual < tolerance){
      std::cout << "# Conjugate Gradient converged !!! \n";
      break;
    }

    updateSolution(ptr);
    updateResidual(ptr);

    dotProduct (ptr, _Residual_, _Residual_, _Mult2_);
    computeBeta(ptr);

    updateConjugateDirection(ptr);
  }

  file.close();

  writeSolution(ptr);
}

void initSolution(ConjugateGradientDataPtrs &ptr) {
  const int nthr  = 128;
  const int nblck = getBlocks(nthr,systemSize);

  initSolutionGPU<<<nblck,nthr>>>(ptr.devicePtr);
  //check("initSolutionGPU failed");
}

void computeResidual(ConjugateGradientDataPtrs &ptr) {
  const int nthr  = 128;
  const int nblck = getBlocks(nthr,systemSize);

  computeResidualGPU<<<nblck,nthr>>>(ptr.devicePtr);
  //check("computeResidualGPU failed");
}

void initConjugateDirection(ConjugateGradientDataPtrs &ptr) {
  const int nthr  = 128;
  const int nblck = getBlocks(nthr,systemSize);

  initConjugateDirectionGPU<<<nblck,nthr>>>(ptr.devicePtr);
  //check("initConjugateDirectionGPU failed");
}

void dotProduct(ConjugateGradientDataPtrs& ptr, ConjugateGradientEnum A, ConjugateGradientEnum B, ConjugateGradientEnum C) {
  // Compute the inner product of vectors A and B. The result must be stored in
  // vector C, which must be accessible from both the GPU and the CPU.
  const int nthr = 128;
  const int nblck = getBlocks(nthr, systemSize);

  // matVec(..);

  // Hint:
  // ptr.hostPtr->get(A) returns pointer to CPU address of A
  // ptr.hostPtr->get(B) returns pointer to CPU address of B
  // ptr.hostPtr->get(C) returns pointer to CPU address of C

  // ptr.deviceData->get(A) returns pointer to CPU address of A
  // ptr.deviceData->get(B) returns pointer to CPU address of B
  // ptr.deviceData->get(C) returns pointer to CPU address of C
}

void matVec(ConjugateGradientDataPtrs& ptr, ConjugateGradientEnum A, ConjugateGradientEnum B) {
  const int nthr  = 128;
  const int nblck = getBlocks(nthr,systemSize);

  matVecGPU<<<nblck,nthr>>>(ptr.deviceData->get(_Diag_), ptr.deviceData->get(_OffDiag_), ptr.deviceData->get(A), ptr.deviceData->get(B));
  //check("matVecGPU failed");
}

void computeAlpha(ConjugateGradientDataPtrs &ptr){
  *(ptr.hostPtr->alpha) = *(ptr.hostPtr->mult1) / *(ptr.hostPtr->mult2);
}

void computeBeta(ConjugateGradientDataPtrs &ptr){
  *(ptr.hostPtr->beta) = *(ptr.hostPtr->mult2) / *(ptr.hostPtr->mult1);
}

void updateSolution(ConjugateGradientDataPtrs &ptr) {
  const int nthr  = 128;
  const int nblck = getBlocks(nthr, systemSize);

  updateSolutionGPU<<<nblck,nthr>>>(ptr.devicePtr, *ptr.hostPtr->alpha);
  //check("updateSolutionGPU failed");
}

void updateResidual(ConjugateGradientDataPtrs &ptr) {
  const int nthr  = 128;
  const int nblck = getBlocks(nthr, systemSize);

  updateResidualGPU<<<nblck,nthr>>>(ptr.devicePtr, *(ptr.hostPtr->alpha));
  //check("updateResidualGPU failed");
}

void updateConjugateDirection(ConjugateGradientDataPtrs &ptr) {
  const int nthr  = 128;
  const int nblck = getBlocks(nthr,systemSize);

  updateConjugateDirectionGPU<<<nblck,nthr>>>(ptr.devicePtr, *(ptr.hostPtr->beta));
  //check("updateConjugateDirectionGPU failed");
}

void writeSolution(ConjugateGradientDataPtrs &ptr) {
  std::ofstream file;
  file.open("cg.res",std::ios::out | std::ios::trunc);

  hipError_t err = hipMemcpy(ptr.hostPtr->solution, ptr.deviceData->solution, systemSize*sizeof(double), hipMemcpyDeviceToHost);
  if (err != hipSuccess) Stop("memory copy failed");

  for (int i=0; i<systemSize; i++){
    file << std::setw(15) << ptr.hostPtr->solution[i] << "\n";
  }

  file.close();
}


// --------------------------------------------------------------------------------------
//                                    K E R N E L S
// --------------------------------------------------------------------------------------
__global__ void getSPD_GPU(ConjugateGradientData *data) {
  const int index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index >= systemSize) return;

  constexpr double h    = 0.05;
  constexpr double T0   = 300.0;
  constexpr double Tinf = 200.0;
  constexpr double Tend = 400.0;
  constexpr double len  =  10.0;

  constexpr double dx   = len / double(systemSize-1);
  constexpr double fac  = h*dx*dx;

  const int nthr  = 128;
  const int nblck = getBlocks(nthr, systemSize);

  // Initialize the solution at each thread of execution
  initSolutionGPU<<<nblck,nthr>>>(data);

  // LHS / RHS :
  double rhs     = Tinf * fac;
  double diag    = 2.   + fac;
  double offDiag = -1.;

  if (index == 0           ) { rhs = T0;   diag = 1.; }
  if (index == systemSize-1) { rhs = Tend; diag = 1.; }

  data->rhs [index] = rhs;
  data->diag[index] = diag;

  if (index < systemSize-1) data->offDiag[index] = offDiag;
}

__global__ void initSolutionGPU(ConjugateGradientData *data) {
  // Initialize solution vector to zero everywhere except for the boundaries
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  constexpr double T0   = 300.0;
  constexpr double Tend = 400.0;

  double sol = 0.0;

  if (index == 0) sol = T0;
  if (index == systemSize - 1) sol = Tend;

  data->solution[index] = sol;
}

__global__ void initConjugateDirectionGPU(ConjugateGradientData *data) {
  // Initialize the conjugate gradient direction
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int nthr  = 128;
  const int nblck = getBlocks(nthr, systemSize);

  // At the first iteration, the conjugate direction is equal to the initial residual: p0 = r0 = b0 - A * x0
  computeResidualGPU<<<nthr,nblck>>>(data);

  data->direction[index] = data->residual[index];
}

__global__ void computeResidualGPU(ConjugateGradientData *data) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int nthr  = 128;
  const int nblck = getBlocks(nthr, systemSize);

  // residual = rhs - A*solution
  matVecGPU<<<nthr, nblck>>>(data->diag, data->offDiag, data->solution, data->projection);

  data->residual[index] = data->rhs[index] - data->projection[index];
}

__global__ void matVecGPU(double *diag, double *offDiag, double *x, double *y) {
  // Compute the product y = A*x
}

__global__ void updateSolutionGPU(ConjugateGradientData *data, const double alpha) {
  // Compute the CG solution
}

__global__ void updateResidualGPU(ConjugateGradientData *data, const double alpha) {
  // Compute the CG residual
}

__global__ void updateConjugateDirectionGPU(ConjugateGradientData *data, const double beta) {
  // Compute the CG direction
}
